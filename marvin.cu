// Please choose a data type to compile
#define DATATYPE 0
#include "marvin.hpp"

using namespace marvin;
using namespace std;

int main(int argc, char **argv){

    if (argc < 3 || argc >10){
        cout<<"Usage:"<<endl;
        cout<<argv[0]<<" train network.json [model1.marvin[,model2.marvin,...]] [snapshot_iteration]"<<endl;
        cout<<"       example: "<<argv[0]<<" train examples/mnist/lenet.json"<<endl;
        cout<<argv[0]<<" test network.json model1.marvin[,model2.marvin,...] response_name1[,name2,...] file_name1.tensor[,name2.tensor,...] [save_every_n_iterations]"<<endl;
        cout<<"       example: "<<argv[0]<<" test examples/mnist/lenet.json examples/mnist/lenet.marvin ip1,conv2 examples/mnist/ip1.tensor,examples/mnist/conv2.tensor"<<endl;
        cout<<argv[0]<<" activate network.json model1.marvin[,model2.marvin,...] response_name_data response_name1[,name2,...] response1_channels[,response2_channels,...] file_prefix topK maxIterations"<<endl;
        cout<<"       example: "<<argv[0]<<" activate examples/mnist/lenet.json examples/mnist/lenet.marvin data conv1,conv2 [0,1,2],[0,1,2,3,4,5] examples/mnist/filters_ 100 20"<<endl;
        return 0;

    }

    cout<< "====================================================================================================================================="<<endl;
    cout<< ">> Hello, World! This is Marvin. I am at a rough estimate thirty billion times more intelligent than you. Let me give you an example."<<endl;
    cout<< "====================================================================================================================================="<<endl;

    if(0==strcmp(argv[1], "train")){

        Solver solver(argv[2]);
        solver.Malloc(Training);
        solver.randInit();
                
        if (argc==3){       
            solver.train();
        }else if (argc==4 || argc==5){

            vector<string> models = getStringVector(argv[3]);
            for (int m=0;m<models.size();++m)   solver.loadWeights(models[m],true);

            if (argc==4){
                solver.train();
            }else{
                solver.train(atoi(argv[4]));
            }
        }else FatalError(__LINE__);
        
        solver.saveWeights(solver.path + ".marvin");
        
    }else if(0==strcmp(argv[1], "test")){

        Net net(argv[2]);
        net.Malloc(Testing);

        vector<string> models = getStringVector(argv[3]);
        for (int m=0;m<models.size();++m)   net.loadWeights(models[m]);

        if (argc>=6){
            int itersPerSave = 0;
            if (argc==7){
                itersPerSave = atoi(argv[6]);
            }
            net.test(getStringVector(argv[4]), getStringVector(argv[5]), itersPerSave);
        }else if (argc==4){
            net.test();
        }else FatalError(__LINE__);

    }else if(0==strcmp(argv[1], "activate")){

        Net net(argv[2]);
        net.Malloc(Testing);
        
        vector<string> models = getStringVector(argv[3]);
        for (int m=0;m<models.size();++m)   net.loadWeights(models[m]);

        net.getTopActivations(argv[4], getStringVector(argv[5]), getIntVectorVector(argv[6]), argv[7], atoi(argv[8]), atoi(argv[9]));
    }

    return 0;
}
